#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <>
#include <hipcub/hipcub.hpp>
#include <vector>

#define BLOCK_SIZE 256
#define MAX_SAMPLES 512

// Debug mode - set to 1 to enable detailed debugging
#define DEBUG_MODE 0

#if DEBUG_MODE
#define DEBUG_PRINTF(fmt, ...) printf("[DEBUG] Thread %d: " fmt "\n", threadIdx.x + blockIdx.x * blockDim.x, ##__VA_ARGS__)
#define DEBUG_CHECK_BOUNDS(ptr, idx, max_size, name)                             \
    if (idx >= max_size || idx < 0)                                              \
    {                                                                            \
        printf("[ERROR] Thread %d: %s bounds violation - idx:%d, max_size:%d\n", \
               threadIdx.x + blockIdx.x * blockDim.x, name, idx, max_size);      \
        return;                                                                  \
    }
#define DEBUG_CHECK_PTR(ptr, name)                           \
    if (!ptr)                                                \
    {                                                        \
        printf("[ERROR] Thread %d: %s is null pointer\n",    \
               threadIdx.x + blockIdx.x * blockDim.x, name); \
        return;                                              \
    }
#else
#define DEBUG_PRINTF(fmt, ...)
#define DEBUG_CHECK_BOUNDS(ptr, idx, max_size, name)
#define DEBUG_CHECK_PTR(ptr, name)
#endif

// Device functions
__device__ __forceinline__ float3 make_float3_from_ptr(const float *ptr)
{
    return make_float3(ptr[0], ptr[1], ptr[2]);
}

__device__ __forceinline__ void atomic_add_float3(float *dst, float3 src)
{
    atomicAdd(&dst[0], src.x);
    atomicAdd(&dst[1], src.y);
    atomicAdd(&dst[2], src.z);
}

__device__ __forceinline__ float grid_sample_2d(
    const float *grid,
    int C, int H, int W,
    float x, float y,
    int c)
{
    DEBUG_CHECK_PTR(grid, "grid_2d");

    // Early bounds check for channel
    if (c >= C || c < 0)
    {
        DEBUG_PRINTF("grid_sample_2d: channel bounds violation c=%d, C=%d", c, C);
        return 0.0f;
    }

    // Optimized coordinate transformation with fused operations
    x = fmaf(x, 2.0f, -1.0f); // 2*x - 1
    y = fmaf(y, 2.0f, -1.0f); // 2*y - 1

    // Convert to grid coordinates with optimized FMA
    float gx = fmaf(x + 1.0f, 0.5f * (W - 1), 0.0f);
    float gy = fmaf(y + 1.0f, 0.5f * (H - 1), 0.0f);

    // Clamp to valid range
    gx = fmaxf(0.0f, fminf(gx, W - 1.0f));
    gy = fmaxf(0.0f, fminf(gy, H - 1.0f));

    // Fast floor with integer conversion
    int x0 = __float2int_rd(gx); // Fast floor
    int y0 = __float2int_rd(gy);
    int x1 = min(x0 + 1, W - 1);
    int y1 = min(y0 + 1, H - 1);

    float wx = gx - x0;
    float wy = gy - y0;

    // Optimized index calculation with base offset
    int base_offset = c * H * W;
    int idx00 = base_offset + y0 * W + x0;
    int idx01 = base_offset + y0 * W + x1;
    int idx10 = base_offset + y1 * W + x0;
    int idx11 = base_offset + y1 * W + x1;

// Bounds checking only in debug mode for performance
#if DEBUG_MODE
    int max_idx = C * H * W - 1;
    if (idx00 > max_idx || idx01 > max_idx || idx10 > max_idx || idx11 > max_idx)
    {
        DEBUG_PRINTF("grid_sample_2d: array bounds violation - max_idx:%d, indices:[%d,%d,%d,%d]",
                     max_idx, idx00, idx01, idx10, idx11);
        return 0.0f;
    }
#endif

    // Load values and perform bilinear interpolation with FMA optimization
    float v00 = grid[idx00];
    float v01 = grid[idx01];
    float v10 = grid[idx10];
    float v11 = grid[idx11];

    // Optimized bilinear interpolation using FMA instructions
    float wx_inv = 1.0f - wx;
    float wy_inv = 1.0f - wy;

    return fmaf(wx_inv * wy_inv, v00,
                fmaf(wx * wy_inv, v01,
                     fmaf(wx_inv * wy, v10, wx * wy * v11)));
}

__device__ __forceinline__ float grid_sample_1d(
    const float *grid,
    int C, int L,
    float x,
    int c)
{
    DEBUG_CHECK_PTR(grid, "grid_1d");

    // Early bounds check for channel
    if (c >= C || c < 0)
    {
        DEBUG_PRINTF("grid_sample_1d: channel bounds violation c=%d, C=%d", c, C);
        return 0.0f;
    }

    // Optimized coordinate transformation
    x = fmaf(x, 2.0f, -1.0f); // 2*x - 1

    // Convert to grid coordinate with FMA
    float gx = fmaf(x + 1.0f, 0.5f * (L - 1), 0.0f);

    // Clamp to valid range
    gx = fmaxf(0.0f, fminf(gx, L - 1.0f));

    // Fast floor and linear interpolation
    int x0 = __float2int_rd(gx); // Fast floor
    int x1 = min(x0 + 1, L - 1);

    float wx = gx - x0;

    int idx0 = c * L + x0;
    int idx1 = c * L + x1;

// Bounds checking only in debug mode for performance
#if DEBUG_MODE
    int max_idx = C * L - 1;
    if (idx0 > max_idx || idx1 > max_idx)
    {
        DEBUG_PRINTF("grid_sample_1d: array bounds violation - max_idx:%d, indices:[%d,%d]",
                     max_idx, idx0, idx1);
        return 0.0f;
    }
#endif

    // Optimized linear interpolation using FMA
    return fmaf(wx, grid[idx1], (1.0f - wx) * grid[idx0]);
}

__device__ __forceinline__ bool ray_aabb_intersect(
    float3 ray_o, float3 ray_d,
    float3 aabb_min, float3 aabb_max,
    float *t_min, float *t_max)
{

    float3 inv_ray_d = make_float3(
        1.0f / (ray_d.x + 1e-6f),
        1.0f / (ray_d.y + 1e-6f),
        1.0f / (ray_d.z + 1e-6f));

    float3 t1 = make_float3(
        (aabb_min.x - ray_o.x) * inv_ray_d.x,
        (aabb_min.y - ray_o.y) * inv_ray_d.y,
        (aabb_min.z - ray_o.z) * inv_ray_d.z);

    float3 t2 = make_float3(
        (aabb_max.x - ray_o.x) * inv_ray_d.x,
        (aabb_max.y - ray_o.y) * inv_ray_d.y,
        (aabb_max.z - ray_o.z) * inv_ray_d.z);

    *t_min = fmaxf(fmaxf(fminf(t1.x, t2.x), fminf(t1.y, t2.y)), fminf(t1.z, t2.z));
    *t_max = fminf(fminf(fmaxf(t1.x, t2.x), fmaxf(t1.y, t2.y)), fmaxf(t1.z, t2.z));

    return *t_max >= *t_min && *t_max > 0.0f;
}

__global__ void fused_ray_render_kernel(
    const float *__restrict__ rays,             // [N, 6] (origin + direction)
    const float *__restrict__ density_planes,   // [3, C_d, H, W]
    const float *__restrict__ density_lines,    // [3, C_d, L, 1]
    const float *__restrict__ app_planes,       // [3, C_a, H, W]
    const float *__restrict__ app_lines,        // [3, C_a, L, 1]
    const float *__restrict__ basis_mat_weight, // [app_dim, total_app_comp]
    const float *__restrict__ basis_mat_bias,   // [app_dim] or nullptr
    const float *__restrict__ aabb,             // [2, 3] (min, max)
    const int *__restrict__ grid_size,          // [3]
    float step_size,
    int n_samples,
    bool white_bg,
    bool is_train,
    float distance_scale,
    float ray_march_weight_thres,
    float density_shift,                         // Add density_shift parameter
    int n_rays,
    int density_n_comp,
    int app_n_comp,
    int app_dim,
    float *__restrict__ rgb_output,   // [N, 3]
    float *__restrict__ depth_output, // [N]
    hiprandState *__restrict__ rand_states)
{

    int ray_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (ray_idx >= n_rays)
        return;

    // Debug: Print kernel parameters for first few threads
    if (ray_idx < 3)
    {
        DEBUG_PRINTF("Kernel params - ray_idx:%d, grid_size:[%d,%d,%d], density_n_comp:%d, app_n_comp:%d, app_dim:%d",
                     ray_idx, grid_size[0], grid_size[1], grid_size[2], density_n_comp, app_n_comp, app_dim);
    }

    // Check critical pointers
    DEBUG_CHECK_PTR(rays, "rays");
    DEBUG_CHECK_PTR(density_planes, "density_planes");
    DEBUG_CHECK_PTR(density_lines, "density_lines");
    DEBUG_CHECK_PTR(app_planes, "app_planes");
    DEBUG_CHECK_PTR(app_lines, "app_lines");
    DEBUG_CHECK_PTR(basis_mat_weight, "basis_mat_weight");
    DEBUG_CHECK_PTR(aabb, "aabb");
    DEBUG_CHECK_PTR(grid_size, "grid_size");
    DEBUG_CHECK_PTR(rgb_output, "rgb_output");
    DEBUG_CHECK_PTR(depth_output, "depth_output");

    // Load ray data
    float3 ray_o = make_float3(rays[ray_idx * 6 + 0], rays[ray_idx * 6 + 1], rays[ray_idx * 6 + 2]);
    float3 ray_d = make_float3(rays[ray_idx * 6 + 3], rays[ray_idx * 6 + 4], rays[ray_idx * 6 + 5]);

    // Load AABB
    float3 aabb_min = make_float3(aabb[0], aabb[1], aabb[2]);
    float3 aabb_max = make_float3(aabb[3], aabb[4], aabb[5]);

    // Ray-AABB intersection
    float t_min, t_max;
    if (!ray_aabb_intersect(ray_o, ray_d, aabb_min, aabb_max, &t_min, &t_max))
    {
        // Ray doesn't intersect AABB
        if (white_bg)
        {
            rgb_output[ray_idx * 3 + 0] = 1.0f;
            rgb_output[ray_idx * 3 + 1] = 1.0f;
            rgb_output[ray_idx * 3 + 2] = 1.0f;
        }
        else
        {
            rgb_output[ray_idx * 3 + 0] = 0.0f;
            rgb_output[ray_idx * 3 + 1] = 0.0f;
            rgb_output[ray_idx * 3 + 2] = 0.0f;
        }
        depth_output[ray_idx] = t_max;
        return;
    }

    t_min = fmaxf(t_min, 0.0f);

    // Sample points along ray with optimized loop structure
    float3 rgb_acc = make_float3(0.0f, 0.0f, 0.0f);
    float alpha_acc = 0.0f;
    float depth_acc = 0.0f;

    hiprandState local_rand_state;
    if (is_train && rand_states)
    {
        local_rand_state = rand_states[ray_idx];
    }

    float step_size_t = step_size;
    const float inv_step_size = 1.0f / step_size_t;

    // Pre-compute grid dimensions for efficiency
    const int H = grid_size[1];
    const int W = grid_size[0];
    const int D = grid_size[2];
    const int total_density_comp = 3 * density_n_comp;
    const int total_app_comp = 3 * app_n_comp;

    // Cache AABB normalization factors for better performance
    const float3 aabb_size = make_float3(
        aabb_max.x - aabb_min.x,
        aabb_max.y - aabb_min.y,
        aabb_max.z - aabb_min.z);
    const float3 inv_aabb_size = make_float3(
        1.0f / aabb_size.x,
        1.0f / aabb_size.y,
        1.0f / aabb_size.z);

    for (int i = 0; i < n_samples; i++)
    {
        float t = t_min + (float)i * step_size_t;
        if (t > t_max)
            break;

        // Add jitter for training with optimized random generation
        if (is_train && rand_states)
        {
            t += hiprand_uniform(&local_rand_state) * step_size_t;
        }

        // Sample point with vectorized computation
        float3 pos = make_float3(
            fmaf(t, ray_d.x, ray_o.x),
            fmaf(t, ray_d.y, ray_o.y),
            fmaf(t, ray_d.z, ray_o.z));

        // Optimized normalization with pre-computed inverse
        float3 norm_pos = make_float3(
            (pos.x - aabb_min.x) * inv_aabb_size.x,
            (pos.y - aabb_min.y) * inv_aabb_size.y,
            (pos.z - aabb_min.z) * inv_aabb_size.z);

        // Early bounds check with optimized comparison
        if (__any_sync(0xffffffff, norm_pos.x < 0.0f || norm_pos.x > 1.0f ||
                                       norm_pos.y < 0.0f || norm_pos.y > 1.0f ||
                                       norm_pos.z < 0.0f || norm_pos.z > 1.0f))
        {
            continue;
        }

        // Compute density feature with optimized memory access
        float sigma_feature = 0.0f;

        // Process density features using optimized concatenated tensor layout
        if (density_planes && density_lines)
        {
            // Pre-load normalized positions for better cache utilization
            const float norm_x = norm_pos.x;
            const float norm_y = norm_pos.y;
            const float norm_z = norm_pos.z;

// XY plane with Z line - unrolled for better performance
#pragma unroll 4
            for (int c = 0; c < density_n_comp; c++)
            {
                float plane_val = grid_sample_2d(density_planes, total_density_comp, H, W, norm_x, norm_y, c);
                float line_val = grid_sample_1d(density_lines, total_density_comp, D, norm_z, c);
                sigma_feature = fmaf(plane_val, line_val, sigma_feature);
            }

            // XZ plane with Y line - offset by density_n_comp
            const int xz_offset = density_n_comp;
#pragma unroll 4
            for (int c = 0; c < density_n_comp; c++)
            {
                int comp_idx = xz_offset + c;
                float plane_val = grid_sample_2d(density_planes, total_density_comp, D, W, norm_x, norm_z, comp_idx);
                float line_val = grid_sample_1d(density_lines, total_density_comp, H, norm_y, comp_idx);
                sigma_feature = fmaf(plane_val, line_val, sigma_feature);
            }

            // YZ plane with X line - offset by 2*density_n_comp
            const int yz_offset = 2 * density_n_comp;
#pragma unroll 4
            for (int c = 0; c < density_n_comp; c++)
            {
                int comp_idx = yz_offset + c;
                float plane_val = grid_sample_2d(density_planes, total_density_comp, D, H, norm_y, norm_z, comp_idx);
                float line_val = grid_sample_1d(density_lines, total_density_comp, W, norm_x, comp_idx);
                sigma_feature = fmaf(plane_val, line_val, sigma_feature);
            }
        }

        // Proper density computation matching PyTorch implementation
        // Apply density_shift and softplus activation like PyTorch
        float shifted_feature = sigma_feature + density_shift; // Use passed density_shift parameter
        float sigma = log1pf(__expf(shifted_feature));         // softplus(x) = log(1 + exp(x))

        // Improved alpha computation with clamped integration
        float dt = step_size_t * distance_scale;
        float sigma_dt = fminf(sigma * dt, 15.0f); // Increased clamp for better dynamic range
        float alpha = 1.0f - __expf(-sigma_dt);    // Fast exponential
        float weight = alpha * (1.0f - alpha_acc);

        // Enhanced early termination for performance
        if (weight < ray_march_weight_thres)
        {
            continue;
        }

        // More aggressive alpha accumulation threshold
        if (alpha_acc > 0.98f)
        {
            break;
        }

        // Compute appearance features with optimized memory access
        if (weight > ray_march_weight_thres)
        {
            // Use shared memory for app features for better performance
            __shared__ float shared_features[BLOCK_SIZE * 16]; // Assuming max 16 features per thread
            float *app_features = &shared_features[threadIdx.x * 16];

// Initialize features efficiently
#pragma unroll 8
            for (int j = 0; j < min(app_dim, 16); j++)
            {
                app_features[j] = 0.0f;
            }

            // Handle remaining features if app_dim > 16
            for (int j = 16; j < app_dim && j < 64; j++)
            {
                app_features[j] = 0.0f;
            }

            // Optimized appearance feature computation with cached values
            const float norm_x = norm_pos.x;
            const float norm_y = norm_pos.y;
            const float norm_z = norm_pos.z;

            // Compute appearance features: process all 3 planes (XY, XZ, YZ) 
            // Each plane contributes app_n_comp features, total = 3 * app_n_comp = total_app_comp
            // Must match PyTorch: concatenate all plane features, then apply basis matrix
            
            // Temporary storage for all combined features (plane * line)
            float combined_features[144]; // Max possible features
            int feature_idx = 0;

            // XY plane with Z line
            for (int c = 0; c < app_n_comp; c++, feature_idx++)
            {
                float plane_val = grid_sample_2d(app_planes, total_app_comp, H, W, norm_x, norm_y, c);
                float line_val = grid_sample_1d(app_lines, total_app_comp, D, norm_z, c);
                combined_features[feature_idx] = plane_val * line_val;
            }

            // XZ plane with Y line
            const int xz_offset = app_n_comp;
            for (int c = 0; c < app_n_comp; c++, feature_idx++)
            {
                int comp_idx = xz_offset + c;
                float plane_val = grid_sample_2d(app_planes, total_app_comp, D, W, norm_x, norm_z, comp_idx);
                float line_val = grid_sample_1d(app_lines, total_app_comp, H, norm_y, comp_idx);
                combined_features[feature_idx] = plane_val * line_val;
            }

            // YZ plane with X line  
            const int yz_offset = 2 * app_n_comp;
            for (int c = 0; c < app_n_comp; c++, feature_idx++)
            {
                int comp_idx = yz_offset + c;
                float plane_val = grid_sample_2d(app_planes, total_app_comp, D, H, norm_y, norm_z, comp_idx);
                float line_val = grid_sample_1d(app_lines, total_app_comp, W, norm_x, comp_idx);
                combined_features[feature_idx] = plane_val * line_val;
            }

            // Now apply basis matrix transformation: combined_features[total_app_comp] -> app_features[app_dim]
            // This matches PyTorch: basis_mat((plane_coef_point * line_coef_point).T)
            for (int d = 0; d < app_dim; d++)
            {
                float result = 0.0f;
                for (int f = 0; f < total_app_comp; f++)
                {
                    int basis_idx = d * total_app_comp + f;
                    result = fmaf(basis_mat_weight[basis_idx], combined_features[f], result);
                }
                app_features[d] = result;
            }

            // Add bias if available
            if (basis_mat_bias)
            {
                for (int d = 0; d < app_dim; d++)
                {
                    app_features[d] += basis_mat_bias[d];
                }
            }

            // Optimized RGB conversion with fast math functions
            float3 rgb;
            if (app_dim >= 3)
            {
                // Enhanced sigmoid with fast math for better performance
                rgb.x = __fdividef(1.0f, 1.0f + __expf(-app_features[0]));
                rgb.y = __fdividef(1.0f, 1.0f + __expf(-app_features[1]));
                rgb.z = __fdividef(1.0f, 1.0f + __expf(-app_features[2]));
            }
            else if (app_dim == 1)
            {
                float gray = fmaxf(0.0f, fminf(1.0f, app_features[0]));
                rgb = make_float3(gray, gray, gray);
            }
            else
            {
                rgb.x = fmaxf(0.0f, fminf(1.0f, app_features[0]));
                rgb.y = app_dim > 1 ? fmaxf(0.0f, fminf(1.0f, app_features[1])) : rgb.x;
                rgb.z = rgb.x;
            }

            // Optimized color accumulation with FMA
            rgb_acc.x = fmaf(weight, rgb.x, rgb_acc.x);
            rgb_acc.y = fmaf(weight, rgb.y, rgb_acc.y);
            rgb_acc.z = fmaf(weight, rgb.z, rgb_acc.z);
            depth_acc = fmaf(weight, t, depth_acc);
        }

        alpha_acc += weight;

        // Early termination if alpha is close to 1
        if (alpha_acc > 0.99f)
            break;
    }

    // Apply background
    if (white_bg)
    {
        rgb_acc.x += (1.0f - alpha_acc);
        rgb_acc.y += (1.0f - alpha_acc);
        rgb_acc.z += (1.0f - alpha_acc);
    }

    // Debug: Check output bounds before writing
    int rgb_idx_r = ray_idx * 3 + 0;
    int rgb_idx_g = ray_idx * 3 + 1;
    int rgb_idx_b = ray_idx * 3 + 2;

    if (ray_idx < 3)
    {
        DEBUG_PRINTF("Final output - ray_idx:%d, rgb_acc:[%.3f,%.3f,%.3f], depth_acc:%.3f, alpha_acc:%.3f",
                     ray_idx, rgb_acc.x, rgb_acc.y, rgb_acc.z, depth_acc, alpha_acc);
        DEBUG_PRINTF("Output indices - rgb:[%d,%d,%d], depth:%d", rgb_idx_r, rgb_idx_g, rgb_idx_b, ray_idx);
    }

    // Check bounds before writing
    DEBUG_CHECK_BOUNDS(rgb_output, rgb_idx_r, n_rays * 3, "rgb_output_r");
    DEBUG_CHECK_BOUNDS(rgb_output, rgb_idx_g, n_rays * 3, "rgb_output_g");
    DEBUG_CHECK_BOUNDS(rgb_output, rgb_idx_b, n_rays * 3, "rgb_output_b");
    DEBUG_CHECK_BOUNDS(depth_output, ray_idx, n_rays, "depth_output");

    // Store results
    rgb_output[ray_idx * 3 + 0] = rgb_acc.x;
    rgb_output[ray_idx * 3 + 1] = rgb_acc.y;
    rgb_output[ray_idx * 3 + 2] = rgb_acc.z;
    depth_output[ray_idx] = depth_acc;

    // Update random state
    if (is_train && rand_states)
    {
        rand_states[ray_idx] = local_rand_state;
    }
}

__global__ void init_curand_kernel(hiprandState *state, unsigned long seed, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        hiprand_init(seed, idx, 0, &state[idx]);
    }
}

std::vector<torch::Tensor> fused_ray_render_cuda_forward(
    torch::Tensor rays,
    torch::Tensor density_planes,
    torch::Tensor density_lines,
    torch::Tensor app_planes,
    torch::Tensor app_lines,
    torch::Tensor basis_mat_weight,
    torch::Tensor basis_mat_bias,
    torch::Tensor aabb,
    torch::Tensor grid_size,
    float step_size,
    int n_samples,
    bool white_bg,
    bool is_train,
    float distance_scale,
    float ray_march_weight_thres,
    float density_shift)
{

    const int n_rays = rays.size(0);
    // FIXED: Correct component calculation for concatenated tensors
    // For TensorVMSplit: density_planes shape [48, H, W] = 3 planes * 16 components each
    // So individual density_n_comp = 48/3 = 16
    const int density_n_comp = density_planes.size(0) / 3; // Individual component count per plane
    const int app_n_comp = app_planes.size(0) / 3;         // Individual component count per plane
    const int app_dim = basis_mat_weight.size(0);

#if DEBUG_MODE
    // Print calculation details for debugging
    printf("[DEBUG] Tensor size calculations:\n");
    printf("  density_planes.size(0) = %ld\n", density_planes.size(0));
    printf("  density_planes.size(0) / 3 = %d\n", (int)(density_planes.size(0) / 3));
    printf("  app_planes.size(0) = %ld\n", app_planes.size(0));
    printf("  app_planes.size(0) / 3 = %d\n", (int)(app_planes.size(0) / 3));
    printf("  Final: density_n_comp=%d, app_n_comp=%d\n", density_n_comp, app_n_comp);
#endif

#if DEBUG_MODE
    // Print tensor dimensions for debugging
    printf("[DEBUG] Forward function input tensor sizes:\n");
    printf("  rays: [%ld, %ld]\n", rays.size(0), rays.size(1));
    printf("  density_planes ndim: %d\n", density_planes.dim());
    if (density_planes.dim() >= 4)
    {
        printf("  density_planes: [%ld, %ld, %ld, %ld]\n", density_planes.size(0), density_planes.size(1), density_planes.size(2), density_planes.size(3));
    }
    else if (density_planes.dim() == 3)
    {
        printf("  density_planes: [%ld, %ld, %ld]\n", density_planes.size(0), density_planes.size(1), density_planes.size(2));
    }
    printf("  density_lines ndim: %d\n", density_lines.dim());
    if (density_lines.dim() >= 3)
    {
        printf("  density_lines: [%ld, %ld, %ld]\n", density_lines.size(0), density_lines.size(1), density_lines.size(2));
    }
    else if (density_lines.dim() == 2)
    {
        printf("  density_lines: [%ld, %ld]\n", density_lines.size(0), density_lines.size(1));
    }
    printf("  app_planes ndim: %d\n", app_planes.dim());
    if (app_planes.dim() >= 4)
    {
        printf("  app_planes: [%ld, %ld, %ld, %ld]\n", app_planes.size(0), app_planes.size(1), app_planes.size(2), app_planes.size(3));
    }
    else if (app_planes.dim() == 3)
    {
        printf("  app_planes: [%ld, %ld, %ld]\n", app_planes.size(0), app_planes.size(1), app_planes.size(2));
    }
    printf("  app_lines ndim: %d\n", app_lines.dim());
    if (app_lines.dim() >= 3)
    {
        printf("  app_lines: [%ld, %ld, %ld]\n", app_lines.size(0), app_lines.size(1), app_lines.size(2));
    }
    else if (app_lines.dim() == 2)
    {
        printf("  app_lines: [%ld, %ld]\n", app_lines.size(0), app_lines.size(1));
    }
    printf("  basis_mat_weight: [%ld, %ld]\n", basis_mat_weight.size(0), basis_mat_weight.size(1));
    printf("  grid_size tensor: [%ld]\n", grid_size.size(0));
    printf("  CORRECTED Computed: n_rays=%d, density_n_comp=%d (total=%ld), app_n_comp=%d (total=%ld), app_dim=%d\n",
           n_rays, density_n_comp, density_planes.size(0), app_n_comp, app_planes.size(0), app_dim);

    // Print grid size values
    auto grid_size_cpu = grid_size.cpu();
    auto grid_size_accessor = grid_size_cpu.accessor<int, 1>();
    printf("  Grid dimensions: [%d, %d, %d]\n",
           grid_size_accessor[0], grid_size_accessor[1], grid_size_accessor[2]);
#endif

    // Allocate output tensors with gradient support
    // Create tensors that inherit gradient requirements from input tensors
    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(rays.device());
    torch::Tensor rgb_output = torch::zeros({n_rays, 3}, options);
    torch::Tensor depth_output = torch::zeros({n_rays}, options);
    
    // Enable gradients if any input tensor requires gradients
    bool requires_grad = rays.requires_grad() || density_planes.requires_grad() || 
                        density_lines.requires_grad() || app_planes.requires_grad() || 
                        app_lines.requires_grad() || basis_mat_weight.requires_grad();
    
    if (requires_grad) {
        rgb_output.requires_grad_(true);
        depth_output.requires_grad_(true);
    }

    // Setup random states for training
    hiprandState *rand_states = nullptr;
    if (is_train)
    {
        hipMalloc(&rand_states, n_rays * sizeof(hiprandState));
        const int blocks_rand = (n_rays + BLOCK_SIZE - 1) / BLOCK_SIZE;
        init_curand_kernel<<<blocks_rand, BLOCK_SIZE>>>(rand_states, time(NULL), n_rays);
        hipDeviceSynchronize();
    }

    // Launch kernel
    const int blocks = (n_rays + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Handle optional bias matrix pointer
    float *bias_ptr;
    if (basis_mat_bias.numel() > 0)
    {
        bias_ptr = basis_mat_bias.data_ptr<float>();
    }
    else
    {
        bias_ptr = nullptr;
    }

    fused_ray_render_kernel<<<blocks, BLOCK_SIZE>>>(
        rays.data_ptr<float>(),
        density_planes.data_ptr<float>(),
        density_lines.data_ptr<float>(),
        app_planes.data_ptr<float>(),
        app_lines.data_ptr<float>(),
        basis_mat_weight.data_ptr<float>(),
        bias_ptr,
        aabb.data_ptr<float>(),
        grid_size.data_ptr<int>(),
        step_size,
        n_samples,
        white_bg,
        is_train,
        distance_scale,
        ray_march_weight_thres,
        density_shift,                    // Pass density_shift to kernel
        n_rays,
        density_n_comp,
        app_n_comp,
        app_dim,
        rgb_output.data_ptr<float>(),
        depth_output.data_ptr<float>(),
        rand_states);

    // Cleanup
    if (rand_states)
    {
        hipFree(rand_states);
    }

    hipDeviceSynchronize();

    // Return RGB and depth as separate tensors in a vector
    return {rgb_output, depth_output};
}
