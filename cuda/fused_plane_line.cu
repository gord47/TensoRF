#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <vector>

__device__ float bilinear_interp(const float* plane, float x, float y, int H, int W) {
    x = fmaxf(0.0f, fminf(x * 0.5f + 0.5f, 1.0f));
    y = fmaxf(0.0f, fminf(y * 0.5f + 0.5f, 1.0f));
    float fx = x * (W - 1);
    float fy = y * (H - 1);
    int x0 = floorf(fx), y0 = floorf(fy);
    int x1 = min(x0 + 1, W - 1), y1 = min(y0 + 1, H - 1);
    float dx = fx - x0, dy = fy - y0;

    float val = 0.0f;
    val += plane[y0 * W + x0] * (1 - dx) * (1 - dy);
    val += plane[y0 * W + x1] * dx * (1 - dy);
    val += plane[y1 * W + x0] * (1 - dx) * dy;
    val += plane[y1 * W + x1] * dx * dy;
    return val;
}

__device__ float linear_interp(const float* line, float z, int L) {
    z = fmaxf(0.0f, fminf(z * 0.5f + 0.5f, 1.0f));
    float fz = z * (L - 1);
    int z0 = floorf(fz), z1 = min(z0 + 1, L - 1);
    float dz = fz - z0;
    return line[z0] * (1 - dz) + line[z1] * dz;
}

__global__ void fused_plane_line_kernel(
    const float* __restrict__ planes,
    const float* __restrict__ lines,
    const float* __restrict__ coord_plane,
    const float* __restrict__ coord_line,
    float* __restrict__ out,
    int C, int H, int W, int L, int N
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    float acc = 0.0f;
    for (int axis = 0; axis < 3; ++axis) {
        for (int c = 0; c < C; ++c) {
            const float* plane = planes + axis * C * H * W + c * H * W;
            const float* line = lines + axis * C * L + c * L;

            float x = coord_plane[axis * N * 2 + i * 2 + 0];
            float y = coord_plane[axis * N * 2 + i * 2 + 1];
            float z = coord_line[axis * N + i];

            float p = bilinear_interp(plane, x, y, H, W);
            float l = linear_interp(line, z, L);
            acc += p * l;
        }
    }
    out[i] = acc;
}

std::vector<torch::Tensor> fused_plane_line_forward_cuda(
    torch::Tensor planes,
    torch::Tensor lines,
    torch::Tensor coord_plane,
    torch::Tensor coord_line
) {
    int C = planes.size(1), H = planes.size(2), W = planes.size(3);
    int L = lines.size(2), N = coord_plane.size(1);

    auto output = torch::zeros({N}, planes.options());

    const int threads = 256;
    const int blocks = (N + threads - 1) / threads;

    fused_plane_line_kernel<<<blocks, threads>>>(
        planes.contiguous().data_ptr<float>(),
        lines.contiguous().data_ptr<float>(),
        coord_plane.contiguous().data_ptr<float>(),
        coord_line.contiguous().data_ptr<float>(),
        output.data_ptr<float>(),
        C, H, W, L, N
    );

    return {output};
}
