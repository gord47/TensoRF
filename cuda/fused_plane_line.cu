#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <vector>

__device__ float bilinear_interp(const float* plane, float x, float y, int H, int W) {
    // grid_sample with align_corners=True: x, y ∈ [-1, 1] → index in [0, size - 1]
    float fx = (x + 1.f) * 0.5f * (W - 1);
    float fy = (y + 1.f) * 0.5f * (H - 1);

    fx = fminf(fmaxf(fx, 0.0f), W - 1.0f);
    fy = fminf(fmaxf(fy, 0.0f), H - 1.0f);

    int x0 = floorf(fx), y0 = floorf(fy);
    int x1 = min(x0 + 1, W - 1), y1 = min(y0 + 1, H - 1);
    float dx = fx - x0, dy = fy - y0;

    float val = 0.0f;
    val += plane[y0 * W + x0] * (1 - dx) * (1 - dy);
    val += plane[y0 * W + x1] * dx * (1 - dy);
    val += plane[y1 * W + x0] * (1 - dx) * dy;
    val += plane[y1 * W + x1] * dx * dy;
    return val;
}

__device__ float linear_interp(const float* line, float z, int L) {
    float fz = (z + 1.f) * 0.5f * (L - 1);
    fz = fminf(fmaxf(fz, 0.0f), L - 1.0f);

    int z0 = floorf(fz), z1 = min(z0 + 1, L - 1);
    float dz = fz - z0;
    return line[z0] * (1 - dz) + line[z1] * dz;
}

__global__ void fused_plane_line_kernel(
    const float* __restrict__ planes,
    const float* __restrict__ lines,
    const float* __restrict__ coord_plane, // [3, N, 2] flattened
    const float* __restrict__ coord_line,  // [3, N] flattened
    float* __restrict__ out,
    int C, int H, int W, int L, int N
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    float acc = 0.0f;
    for (int axis = 0; axis < 3; ++axis) {
        for (int c = 0; c < C; ++c) {
            const float* plane = planes + axis * C * H * W + c * H * W;
            const float* line  = lines + axis * C * L + c * L;

            float x = coord_plane[(axis * N + i) * 2 + 0];
            float y = coord_plane[(axis * N + i) * 2 + 1];
            float z = coord_line[axis * N + i];

            float p = bilinear_interp(plane, x, y, H, W);
            float l = linear_interp(line, z, L);
            acc += p * l;
        }
    }
    out[i] = acc;
}

std::vector<torch::Tensor> fused_plane_line_forward_cuda(
    torch::Tensor planes,       // [3, C, H, W]
    torch::Tensor lines,        // [3, C, L]
    torch::Tensor coord_plane,  // [3, N, 2]
    torch::Tensor coord_line    // [3, N]
) {
    int C = planes.size(1), H = planes.size(2), W = planes.size(3);
    int L = lines.size(2), N = coord_plane.size(1);

    auto output = torch::zeros({N}, planes.options());

    const int threads = 256;
    const int blocks = (N + threads - 1) / threads;

    fused_plane_line_kernel<<<blocks, threads>>>(
        planes.contiguous().data_ptr<float>(),
        lines.contiguous().data_ptr<float>(),
        coord_plane.contiguous().data_ptr<float>(),
        coord_line.contiguous().data_ptr<float>(),
        output.data_ptr<float>(),
        C, H, W, L, N
    );

    return {output};
}
